#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <sstream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <math.h>


int* readfile(const char* filename, int* size);
/*****************************************************
while !stable
	propose in parallel
	block
	accept/reject in parallel
	block
*****************************************************/

__device__ bool stable;
__device__ bool no_match;
__device__ bool gpu_reduced_size_empty;

__global__ void p1_proposal(int* preference_lists, int* proposal_to, int* proposed_to, int N) {

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	stable = true;

	if (row < N)
	{
		__syncthreads();
		if (proposed_to[row] >= (N-1))
		{
			no_match = true;
			return;
		}
		
		// if proposal was rejected, or havent proposed to anyone yet
		if (proposal_to[row] == N)
		{
			proposal_to[row] = preference_lists[row * N + proposed_to[row] + 1];
		}
	}
}

__global__ void p1_rejection(int* preference_lists, int* proposal_to, int* proposal_from, int* proposed_to, int N) {

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int proposee;

	if (row < N)
	{
		__syncthreads();

		proposee = proposal_to[row];
		
		int op = N;
		for (int i = 0; i < (N-1); i++)
		{
			if (preference_lists[proposee * N + i + 1] == row)
			{
				op = i;
				break;
			}
		}

		if (op == N)
		{
			no_match = true;
			return;
		}

		int op_curr = N;
		for (int i = 0; i < (N-1); i++)
		{
			if (preference_lists[proposee * N + i + 1] == proposal_from[proposee*N])
			{
				
				op_curr = i;
				break;
			}
		}

		if (op < op_curr)
		{
			if (proposal_from[proposee*N] != N)
			{
				proposal_to[proposal_from[proposee*N]] = N;
				stable = false;
			}
			proposal_from[proposee*N + op + 1] = row;
		}
		else if (op == op_curr)
		{
		}
		else
		{
			stable = false;
		}
		atomicAdd(&proposed_to[row], 1);

		__syncthreads();
	}
}

__global__ void p1_accept(int* proposal_from, int* proposal_to, int* C, int* rank, int N) {

	int i = threadIdx.x;
	int j = blockIdx.x;
	
	if (i < N && j < N)
	{
		C[j * N + i] = N;

		if (proposal_from[j * N + i] != N)
		{
			C[j * N + i] = rank[j * N + proposal_from[j * N + i]];
		}

		for (int d = 1; d < N; d *= 2)
		{
			if (i - d >= 0)
			{
				if (C[j * N + i] > C[j * N + i - d])
				{
					C[j * N + i] = C[j * N + i - d];
				}
			}
			__syncthreads();
		}

		if (C[j * N + i] != N)
		{
			proposal_from[j * N] = proposal_from[j * N + i];
		}
		if (i != 0)
		{
			proposal_from[j * N + i] = N;
		}
	}
}

__global__ void p1_evaluate(int* proposal_from, int* proposal_to, int N)
{
	int i = threadIdx.x;

	if (i < N)
	{
		proposal_to[i] = N;

		__syncthreads();

		if (proposal_from[i * N] != N) {
			proposal_to[proposal_from[i * N]] = i;
		}
		else {
			stable = false;
		}

		__syncthreads();
	}
}

__global__ void p1_remove(int* preference_lists, int* proposal_from, int* rank, int N)
{
	int i = threadIdx.x;
	int j = blockIdx.x;

	if (i < (N - 1) && j < N)
	{
		if (rank[j * N + i] > rank[j * N + proposal_from[j * N]] && rank[j * N + i] != N)
		{
			preference_lists[j * N + rank[j * N + i]] = N;
			preference_lists[i * N + rank[i * N + j]] = N;
		}
	}
}

__global__ void p1_shift(int* preference_lists, int* reduced_size, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		int count = 0;

		for (int d = 0; d < N; d++)
		{
			if (preference_lists[i * N + d] != N)
			{
				preference_lists[i * N + count++] = preference_lists[i * N + d];
			}
		}

		reduced_size[i] = count - 1;
		if (reduced_size[i] <= 0)
		{
			no_match = true;
			return;
		}

		while (count < N)
		{
			preference_lists[i * N + count++] = N;
		}
	}
}

__global__ void get_rank(int* preference_lists, int N, int* rank, int* reduced_size)
 {
	int i = threadIdx.x;
	int j = blockIdx.x;

	if (i < N && j < N)
	{
		rank[j * N + i] = N;
		if (i < reduced_size[j] + 1) {
			rank[j * N + preference_lists[j * N + i]] = i;
		}
		if (i == j)
		{
			rank[j * N + i] = N;
		}
	}
	__syncthreads();
		
}
__global__ void p2_remove(int* preference_lists, int* reduced_size, int* rotations, int N, int* rank, int count)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	__syncthreads();
	if (i < count)
	{
		if (rotations[i] != N)
		{
			if (i % 2 == 1)//odd
			{
				preference_lists[rotations[i] * N + rank[rotations[i] * N + rotations[i - 1]]] = N;
				atomicSub(&reduced_size[rotations[i]], 1);

				if (reduced_size[rotations[i]] == 0)
				{
					no_match = true;
					return;
				}
			}
			else //even
			{
				preference_lists[rotations[i] * N + rank[rotations[i] * N + rotations[i + 1]]] = N;
				atomicSub(&reduced_size[rotations[i]], 1);

				if (reduced_size[rotations[i]] == 0)
				{
					no_match = true;
					return;
				}
			}
		}
	}
}


__global__ void p2_getSecondLastChoice(int* preference_lists, int* last_choice, int* second_choice, int* reduced_size, int N)
{
	int i = threadIdx.x;
	int j = blockIdx.x;

	if (i < N && j < N)
	{
		if (preference_lists[j * N + i] < N && reduced_size[j] > 1)
		{
			// second choice
			if (i == 2)
			{
				second_choice[j] = preference_lists[j * N + i];
			}

			// last choice needs to be size
			if (i == reduced_size[j])
			{
				last_choice[j] = preference_lists[j * N + i];
			}
		}
		else if (reduced_size[j] == 0)
		{
			no_match = true;
			return;
		}
	}

}

__global__ void get_reduced_sizes(int* reduced_size, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		if (reduced_size[i] <= 0)
		{
			gpu_reduced_size_empty = true;
		}
	}
}

__global__ void fill_matching_zeros(int* matching, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		matching[i] = 0;
	}
}

__global__ void fill_matching(int* preference_lists, int* matching, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		matching[i] = preference_lists[i * N + 1];
	}
}

std::vector<int> stable_roommate(std::vector<std::vector<int>> preference_lists_vector, int N) {

	int NUM_BLOCKS;
	int NUM_THREADS;

	if (N % 32)
	{
		NUM_BLOCKS = N + (32 - N % 32);
		NUM_THREADS = N + (32 - N % 32);
	}
	else
	{
		NUM_BLOCKS = N;
		NUM_THREADS = N;
	}

	int* gpu_preference_lists;
	int* gpu_proposal_to;
	int* gpu_proposal_from;
	int* gpu_proposed_to;
	int* gpu_matching;
	int* gpu_reduced_size;
	int* gpu_second_choice;
	int* gpu_last_choice;
	int* gpu_rotations;
	int* gpu_C;

	int* preference_lists = new int[N*N];

	for (int i = 0; i < N; i++)
	{
		preference_lists[i*N] = i;
		for (int j = 1; j < N; j++)
		{
			preference_lists[i*N+j] = preference_lists_vector[i][j-1];
		}
	}

	int* proposal_from = new int[N*N];
	proposal_from = (int *)calloc(N*N, sizeof(*proposal_from));
	std::replace(proposal_from, proposal_from + N*N, 0, N);

	int *proposed_to = new int[N];
	proposed_to = (int *)calloc(N, sizeof(*proposed_to));
	int *proposal_to = new int[N];
	proposal_to = (int *)calloc(N, sizeof(*proposal_to));
	std::replace(proposal_to, proposal_to + N, 0, N);
	int* reduced_size = new int[N];
	reduced_size = (int*)calloc(N, sizeof(*reduced_size));
	std::replace(reduced_size, reduced_size + N, 0, N-1);
	int* second_choice = new int[N];
	second_choice = (int*)calloc(N, sizeof(*second_choice));
	std::replace(second_choice, second_choice + N, 0, N);
	int* last_choice = new int[N];
	last_choice = (int*)calloc(N, sizeof(*last_choice));
	std::replace(last_choice, last_choice + N, 0, N);
	int* rotations = new int[N * N];
	rotations = (int*)calloc(N * N, sizeof(*rotations));
	std::replace(rotations, rotations + N * N, 0, N);
	int* C = new int[N * N];
	C = (int*)calloc(N * N, sizeof(*C));
	std::replace(C, C + N * N, 0, N);

	int *matching = new int[N];
	matching = (int *)calloc(N, sizeof(*matching));

	hipMalloc(&gpu_preference_lists, N * N * sizeof(int));
	hipMalloc(&gpu_proposal_to, N * sizeof(int));
	hipMalloc(&gpu_proposal_from, N * N * sizeof(int));
	hipMalloc(&gpu_proposed_to, N * sizeof(int));
	hipMalloc(&gpu_matching, N * sizeof(int));
	hipMalloc(&gpu_reduced_size, N * sizeof(int));
	hipMalloc(&gpu_second_choice, N * sizeof(int));
	hipMalloc(&gpu_last_choice, N * sizeof(int));
	hipMalloc(&gpu_rotations, N * N * sizeof(int)); 
	hipMalloc(&gpu_C, N * N * sizeof(int));
	
	bool stable_host;
	bool no_match_host;
	bool reduced_size_empty;

	stable_host = false;
	no_match_host = false;
	hipMemcpy(gpu_preference_lists, preference_lists, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_proposal_to, proposal_to, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_proposal_from, proposal_from, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_proposed_to, proposed_to, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_C, C, N * N * sizeof(int), hipMemcpyHostToDevice);

	int* rank;
	hipMalloc(&rank, N * N * sizeof(int));
	hipMemcpy(gpu_reduced_size, reduced_size, N * sizeof(int), hipMemcpyHostToDevice);
	get_rank <<<NUM_BLOCKS, NUM_THREADS >>> (gpu_preference_lists, N, rank, gpu_reduced_size);

	while (!(stable_host) && !(no_match_host))
	{
		p1_proposal<<<1, NUM_THREADS>>> (gpu_preference_lists, gpu_proposal_to, gpu_proposed_to, N);
		hipDeviceSynchronize();

		p1_rejection<<<1, NUM_THREADS>>> (gpu_preference_lists, gpu_proposal_to, gpu_proposal_from, gpu_proposed_to, N);
		hipDeviceSynchronize();

		p1_accept<<<NUM_BLOCKS, NUM_THREADS>>> (gpu_proposal_from, gpu_proposal_to, gpu_C, rank, N);
		hipDeviceSynchronize();

		p1_evaluate <<<1, NUM_THREADS>>> (gpu_proposal_from, gpu_proposal_to, N);

		hipMemcpyFromSymbol(&stable_host, HIP_SYMBOL(stable), sizeof(stable_host), 0, hipMemcpyDeviceToHost);
		hipMemcpyFromSymbol(&no_match_host, HIP_SYMBOL(no_match), sizeof(no_match_host), 0, hipMemcpyDeviceToHost);
	}

	if (!no_match_host)
	{
		int* rank;
		hipMalloc(&rank, N * N * sizeof(int));
		hipMemcpy(gpu_reduced_size, reduced_size, N * sizeof(int), hipMemcpyHostToDevice);
		get_rank <<<NUM_BLOCKS, NUM_THREADS>>> (gpu_preference_lists, N, rank, gpu_reduced_size);
		p1_remove <<<NUM_BLOCKS, NUM_THREADS>>> (gpu_preference_lists, gpu_proposal_from, rank, N);
		hipDeviceSynchronize();
		hipMemcpy(gpu_reduced_size, reduced_size, N * sizeof(int), hipMemcpyHostToDevice);
		p1_shift <<<1, NUM_THREADS>>> (gpu_preference_lists, gpu_reduced_size, N);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&no_match_host, HIP_SYMBOL(no_match), sizeof(no_match_host), 0, hipMemcpyDeviceToHost);
		p2_getSecondLastChoice <<<NUM_BLOCKS, NUM_THREADS>>> (gpu_preference_lists, gpu_last_choice, gpu_second_choice, gpu_reduced_size, N);
		hipDeviceSynchronize();
	}

	stable_host = false;

	// get rotations
	while (!(stable_host) && !(no_match_host))
	{
		stable_host = true;

		for (int i = 0; i < N; i++)
		{
			hipMemcpy(reduced_size, gpu_reduced_size, N * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(second_choice, gpu_second_choice, N * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(last_choice, gpu_last_choice, N * sizeof(int), hipMemcpyDeviceToHost);
			rotations = (int*)calloc(N * N, sizeof(*rotations));
			std::replace(rotations, rotations + N * N, 0, N);

			if (reduced_size[i] > 1)
			{
				stable_host = false;
				int counter = 0;
				bool cycle_complete = false;
				rotations[counter] = second_choice[i];
				counter++;
				while (!cycle_complete)
				{
					if (counter % 2 == 0)
					{
						rotations[counter] = second_choice[rotations[counter-1]];
					}
					else
					{
						rotations[counter] = last_choice[rotations[counter-1]];
					}

					if (rotations[counter] == i)
					{
						cycle_complete = true;
					}
					counter++;
				}
				
				int* rank;
				int num_threads_counter;

				if (counter % 32)
				{
					num_threads_counter = counter + (32 - counter % 32);
				}
				else
				{
					num_threads_counter = counter;
				}

				hipMalloc(&rank, N * N * sizeof(int));
				hipMemcpy(gpu_rotations, rotations, N * N * sizeof(int), hipMemcpyHostToDevice);
				get_rank <<<NUM_BLOCKS, NUM_THREADS>>> (gpu_preference_lists, N, rank, gpu_reduced_size);
				hipDeviceSynchronize();
				p2_remove <<<1, num_threads_counter>>> (gpu_preference_lists, gpu_reduced_size, gpu_rotations, N, rank, counter);
				hipDeviceSynchronize();
				p1_shift <<<1, NUM_THREADS>>> (gpu_preference_lists, gpu_reduced_size, N);
				hipDeviceSynchronize();
				p2_getSecondLastChoice <<<NUM_BLOCKS, NUM_THREADS>>> (gpu_preference_lists, gpu_last_choice, gpu_second_choice, gpu_reduced_size, N);
				hipDeviceSynchronize();
				hipMemcpyFromSymbol(&no_match_host, HIP_SYMBOL(no_match), sizeof(no_match_host), 0, hipMemcpyDeviceToHost);
			}
		}
	}

	reduced_size_empty = false;

	get_reduced_sizes <<<1, NUM_THREADS>>> (gpu_reduced_size, N);
	hipMemcpyFromSymbol(&reduced_size_empty, HIP_SYMBOL(gpu_reduced_size_empty), sizeof(reduced_size_empty), 0, hipMemcpyDeviceToHost);

	hipMemcpy(gpu_matching, matching, N * sizeof(int), hipMemcpyHostToDevice);

	if (no_match_host || reduced_size_empty)
	{
		fill_matching_zeros <<<1, NUM_THREADS>>> (gpu_matching, N);
	}
	else
	{
		fill_matching <<<1, NUM_THREADS>>> (gpu_preference_lists, gpu_matching, N);
	}
	
	hipDeviceSynchronize();
	hipMemcpy(matching, gpu_matching, N * sizeof(int), hipMemcpyDeviceToHost);

	std::vector<int> matching_vector(matching, matching + N);

	hipFree(gpu_preference_lists);
	hipFree(gpu_proposal_to);
	hipFree(gpu_proposal_from);
	hipFree(gpu_proposed_to);
	hipFree(gpu_matching);
	hipFree(gpu_reduced_size);
	hipFree(gpu_second_choice);
	hipFree(gpu_last_choice);
	hipFree(gpu_rotations);

	return matching_vector;
}

int main()
{
	// 2d vector for the preference lists
	std::vector<std::vector<int>> preference_lists;
	int N = 0;
	std::vector<int> matching;

	// input file
	std::ifstream f("inp.txt");
	// get line
	std::string line;

	// while another line to get
	while (std::getline(f, line))
	{
		// inner vector
		std::vector<int> row;
		std::stringstream ss(line);
		std::string data;
		// numbers are separated by commas
		while (std::getline(ss, data, ','))
		{
			// put numbers in vector
			row.push_back(std::stoi(data));
		}
		// put vector in 2d vector
		preference_lists.push_back(row);
		N++;
	}

	matching = stable_roommate(preference_lists, N);

	// output to file
	std::fstream file;
	file.open("outp_p.txt", std::ios::out);

	// if all 0s, no matches. fill with zeros
	if (std::adjacent_find(matching.begin(), matching.end(), std::not_equal_to<>()) == matching.end())
	{
		// print results to text file
		file << "NULL" << "\n";
	}
	else
	{
		for (int i = 0; i < matching.size(); i++)
		{
			file << matching[i] << "\n";
		}
	}
	file.close();

	return 0;
}